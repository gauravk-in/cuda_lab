#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Winter Semester 2013/2014, March 3 - April 4
// ###
// ###
// ### Evgeny Strekalovskiy, Maria Klodt, Jan Stuehmer, Mohamed Souiai
// ###
// ###
// ###



// ###
// ###
// ### TODO: For every student of your group, please provide here:
// ###
// ### name, email, login username (for example p123)
// ###
// ###


#include "aux.h"
#include <iostream>
using namespace std;

// uncomment to use the camera
//#define CAMERA

template<typename T>
__device__ __host__ T min(T a, T b)
{
    return (a < b) ? a : b;
}

template<typename T>
__device__ __host__ T max(T a, T b)
{
    return (a > b) ? a : b;
}

template<typename T>
__device__ __host__ T clamp(T m, T x, T M)
{
    return max(m, min(x, M));
}


__global__ void gradient(float *image, float *vx, float *vy, int w, int h, int nc)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int c = threadIdx.z + blockDim.z * blockIdx.z;
    if (x < w && y < h && c < nc) {
        int i = x + w*y + w*h*c;

        if (x == w-1)
            vx[i] = 0;
        else
            vx[i] = image[i + 1] - image[i];

        if (y == h-1)
            vy[i] = 0;
        else
            vy[i] = image[i + w] - image[i];
    }
}

__device__ __host__ float huber(float s, float epsilon)
{
    return 1.0F / max(epsilon, s);
}

__global__ void compute_P(float *vx, float *vy, int w, int h, int nc, float epsilon)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x < w && y < h) {
        float g = 0;
        for (int c = 0; c < nc; c++) {
            float ux = vx[x + y*w + w*h*c];
            float uy = vy[x + y*w + w*h*c];
            g += ux*ux + uy*uy;
        }
        g = huber(sqrtf(g), epsilon);

        for (int c = 0; c < nc; c++) {
            vx[x + y*w + w*h*c] *= g;
            vy[x + y*w + w*h*c] *= g;
        }
    }
}

__global__ void divergence(float *u1, float *u2, float *div, int w, int h, int nc)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int c = threadIdx.z + blockDim.z * blockIdx.z;
    if (x < w && y < h && c < nc) {
        int i = x + w*y + w*h*c;
        float dx_u1 = u1[i] - ((x > 0) ? u1[i - 1] : 0);
        float dy_u2 = u2[i] - ((y > 0) ? u2[i - w] : 0);
        div[i] = dx_u1 + dy_u2;
    }
}

__global__ void update(float *image, float *dir, int w, int h, int nc, float tau)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int c = threadIdx.z + blockDim.z * blockIdx.z;
    if (x < w && y < h && c < nc) {
        int i = x + w*y + w*h*c;
        image[i] += tau * dir[i];
    }
}

inline int div_ceil(int n, int b) { return (n + b - 1) / b; }

inline dim3 make_grid(dim3 whole, dim3 block)
{
    return dim3(div_ceil(whole.x, block.x),
                div_ceil(whole.y, block.y),
                div_ceil(whole.z, block.z));
}


int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    
    float epsilon = 0.01;
    getParam("epsilon", epsilon, argc, argv);
    cout << "ε: " << epsilon << endl;

    float tau = 0.2 / huber(0, epsilon);
    getParam("tau", tau, argc, argv);
    cout << "τ: " << tau << endl;

    int N = 60;
    getParam("N", N, argc, argv);
    cout << "N: " << N << endl;

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;




    // Set the output image format
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    // ### Define your own output images here as needed




    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn  = new float[(size_t)w*h*nc];
    size_t imageBytes = (size_t)w*h*nc*sizeof(float);

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];




    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);

    dim3 dim(w, h, nc);
    dim3 block2d(32, 16);
    dim3 block3d(16, 8, 3);

    Timer timer; timer.start();
    float *d_image, *d_vx, *d_vy, *d_div;
    hipMalloc(&d_image, imageBytes);
    hipMalloc(&d_vx, imageBytes);
    hipMalloc(&d_vy, imageBytes);
    hipMalloc(&d_div, imageBytes);
    hipMemcpy(d_image, imgIn, imageBytes, hipMemcpyHostToDevice);

    for (int n = 0; n < N; n++) {
        gradient<<< make_grid(dim, block3d), block3d >>>(d_image, d_vx, d_vy, w, h, nc);
        compute_P<<< make_grid(dim, block2d), block2d >>>(d_vx, d_vy, w, h, nc, epsilon);
        divergence<<< make_grid(dim, block3d), block3d >>>(d_vx, d_vy, d_div, w, h, nc);
        update<<< make_grid(dim, block3d), block3d >>>(d_image, d_div, w, h, nc, tau);
    }

    hipMemcpy(imgOut, d_image, imageBytes, hipMemcpyDeviceToHost);
    hipFree(d_image);
    hipFree(d_vx);
    hipFree(d_vy);
    hipFree(d_div);
    timer.end();  float t = timer.get();  // elapsed time in seconds
    cout << "time: " << t*1000 << " ms" << endl;






    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, imgOut);
    showImage("Output", mOut, 100+w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif




    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



