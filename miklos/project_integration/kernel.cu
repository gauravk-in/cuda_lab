#include "hip/hip_runtime.h"
#include "kernel.h"
#include <algorithm>

template<typename T>
__device__ __host__ T min(T a, T b)
{
    return (a < b) ? a : b;
}

template<typename T>
__device__ __host__ T max(T a, T b)
{
    return (a > b) ? a : b;
}

template<typename T>
__device__ __host__ T clamp(T m, T x, T M)
{
    return max(m, min(x, M));
}


__global__ void calculate_F(float *U, float *F, int w, int h, float c1, float c2, float lambda)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x < w && y < h) {
        size_t i = x + (size_t)w*y;
        F[i] = lambda * ((c1 - U[i])*(c1 - U[i]) - (c2 - U[i])*(c2 - U[i]));
    }
}

__device__ float diff_i(float *M, int w, int h, int x, int y)
{
    size_t i = x + (size_t)w*y;
    return (x+1 < w) ? (M[i + 1] - M[i]) : 0.f;
}

__device__ float diff_j(float *M, int w, int h, int x, int y)
{
    size_t i = x + (size_t)w*y;
    return (y+1 < h) ? (M[i + w] - M[i]) : 0.f;
}

__global__ void update_Xij(float *Xi, float *Xj, float *T, float *U, int w, int h, float sigma)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x < w && y < h) {
        size_t i = x + (size_t)w*y;
        float xi = Xi[i] - sigma * (2 * diff_i(U, w, h, x, y) - diff_i(T, w, h, x, y));
        float xj = Xj[i] - sigma * (2 * diff_j(U, w, h, x, y) - diff_j(T, w, h, x, y));
        float dn = max(1.f, sqrtf(xi*xi + xj*xj));
        Xi[i] = xi / dn;
        Xj[i] = xj / dn;
    }
}

__device__ float divergence(float *X, float *Y, int w, int h, int x, int y)
{
    size_t i = x + (size_t)w*y;
    float dx_x = ((x+1 < w) ? X[i] : 0.f) - ((x > 0) ? X[i - 1] : 0.f);
    float dy_y = ((y+1 < h) ? Y[i] : 0.f) - ((y > 0) ? Y[i - w] : 0.f);
    return dx_x + dy_y;
}

__global__ void update_U(float *T, float *Xi, float *Xj, float *F, float *U, int w, int h, float tau)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x < w && y < h) {
        size_t i = x + (size_t)w*y;
        U[i] = clamp(0.f, T[i] - tau * (divergence(Xi, Xj, w, h, x, y) + F[i]), 1.f);
    }
}

__global__ void update_Output(uchar4* output, float *U, int w, int h) {

    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    size_t i = x + (size_t)w*y;
    size_t idx = x + (size_t) w*(h-1 - y);
    unsigned char temp_res = roundf((U[i] * 255.f));
    output[idx].x = temp_res;
    output[idx].y = temp_res;
    output[idx].z = temp_res;
    output[idx].w = 255;

}

inline int div_ceil(int n, int b) { return (n + b - 1) / b; }


__global__ void createVertices(float *in, uchar4* pixel, int w, int h)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned char intensity = roundf(255 * in[y * w + x]);

    // Write positions
    size_t i = x + w*(h-1 - y);
    pixel[i].x = intensity;
    pixel[i].y = intensity;
    pixel[i].z = intensity;
    pixel[i].w = 255;
}

void executeKernel(void *d_in, void *d_out, size_t w, size_t h)
{
    float *d_U = reinterpret_cast<float *>(d_in);
    uchar4 *pixel = reinterpret_cast<uchar4 *>(d_out);

    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid(w / dimBlock.x, h / dimBlock.y, 1);

    // set parameters manually here
    float lambda = 1.0;
    float sigma = 0.4;
    float tau = 0.4;
    int N = 160;
    float c1 = 1.0;
    float c2 = 0.00;

    float *d_T, *d_F, *d_Xi, *d_Xj;
    size_t imageBytes = w*h*sizeof(float);
    hipMalloc(&d_T, imageBytes);
    hipMalloc(&d_F, imageBytes);
    hipMalloc(&d_Xi, imageBytes);
    hipMalloc(&d_Xj, imageBytes);
    hipMemcpy(d_T, d_U, imageBytes, hipMemcpyDeviceToDevice);
    hipMemset(d_Xi, 0, imageBytes);
    hipMemset(d_Xj, 0, imageBytes);

    calculate_F<<< dimGrid, dimBlock >>>(d_U, d_F, w, h, c1, c2, lambda);

    for (int n = 0; n < N; n++) {
        update_Xij<<< dimGrid, dimBlock >>>(d_Xi, d_Xj, d_T, d_U, w, h, sigma);
        std::swap(d_U, d_T);
        update_U<<< dimGrid, dimBlock >>>(d_T, d_Xi, d_Xj, d_F, d_U, w, h, tau);
    }
    update_Output<<< dimGrid, dimBlock >>>(pixel, d_U, w, h);
    hipFree(d_T);
    hipFree(d_F);
    hipFree(d_Xi);
    hipFree(d_Xj);

//    createVertices<<<dimGrid, dimBlock>>>(in, pixel, w, h);
}
