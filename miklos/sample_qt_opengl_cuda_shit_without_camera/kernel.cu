#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void createVertices(float4* positions, float time, unsigned int width, unsigned int height)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate uv coordinates
    float u = x / (float)width;
    float v = y / (float)height;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u * freq + time)
        * cosf(v * freq + time) * 0.5f;

    // Write positions
    positions[y * width + x] = make_float4(u, w, v, 1.0f);
}

void executeKernel(void *positions_, int width, int height, float time)
{
    float4 *positions = (float4 *)positions_;
    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);
    createVertices<<<dimGrid, dimBlock>>>(positions, time, width, height);
}
