#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Winter Semester 2013/2014, March 3 - April 4
// ###
// ###
// ### Evgeny Strekalovskiy, Maria Klodt, Jan Stuehmer, Mohamed Souiai
// ###
// ###
// ###



// ###
// ###
// ### Miklos Homolya, miklos.homolya@tum.de, p056 
// ### Ravikishore Kommajosyula, r.kommajosyula@tum.de, p057
// ### Gaurav Kukreja, gaurav.kukreja@tum.de, p058
// ###
// ###


#include "aux.h"
#include <iostream>
using namespace std;

// uncomment to use the camera
//#define CAMERA

template<typename T>
__device__ __host__ T min(T a, T b)
{
    return (a < b) ? a : b;
}

template<typename T>
__device__ __host__ T max(T a, T b)
{
    return (a > b) ? a : b;
}

template<typename T>
__device__ __host__ T clamp(T m, T x, T M)
{
    return max(m, min(x, M));
}


__device__ __host__ float huber(float s, float epsilon)
{
    return 1.0F / max(epsilon, s);
    //return 1.0F;
    //return expf(-s*s / epsilon) / epsilon;
}

__global__ void diffusivity(float *U, float *G, int w, int h, int nc, float epsilon)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x < w && y < h) {
        float g2 = 0;
        for (int c = 0; c < nc; c++) {
            int i = x + (size_t)w*y + (size_t)w*h*c;
            float ux = ((x < w-1) ? (U[i + 1] - U[i]) : 0);
            float uy = ((y < h-1) ? (U[i + w] - U[i]) : 0);
            g2 += ux*ux + uy*uy;
        }
        G[x + (size_t)w*y] = huber(sqrtf(g2), epsilon);
    }
}

__global__ void sor_update(float *U, float *F, float *G, int w, int h, int nc,
                           float lambda, int color)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x < w && y < h && ((x + y) % 2) == color) {
        int i = x + (size_t)w*y;

        float g_r = ((x+1 < w) ? G[i + 1] : 0);
        float g_l = ((x   > 0) ? G[i - 1] : 0);
        float g_u = ((y+1 < h) ? G[i + w] : 0);
        float g_d = ((y   > 0) ? G[i - w] : 0);
        float g = g_r + g_l + g_u + g_d;

        for (int c = 0; c < nc; c++) {
            int j = x + (size_t)w*y + (size_t)w*h*c;

            float gu = 0.0;
            if (g_r)
                gu += g_r * U[j + 1];
            if (g_l)
                gu += g_l * U[j - 1];
            if (g_u)
                gu += g_u * U[j + w];
            if (g_d)
                gu += g_d * U[j - w];

            U[j] = (2 * F[j] + lambda * gu) / (2 + lambda * g);
        }
    }
}

inline int div_ceil(int n, int b) { return (n + b - 1) / b; }

inline dim3 make_grid(dim3 whole, dim3 block)
{
    return dim3(div_ceil(whole.x, block.x),
                div_ceil(whole.y, block.y),
                div_ceil(whole.z, block.z));
}


int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    
    float sigma = 0.1;
    getParam("sigma", sigma, argc, argv);
    cout << "σ: " << sigma << endl;

    float epsilon = 0.01;
    getParam("epsilon", epsilon, argc, argv);
    cout << "ε: " << epsilon << endl;

    float lambda = 0.25;
    getParam("lambda", lambda, argc, argv);
    cout << "λ: " << lambda << endl;

    int N = 60;
    getParam("N", N, argc, argv);
    cout << "N: " << N << endl;

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }

#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;

    // add noise
    addNoise(mIn, sigma);


    // Set the output image format
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    // ### Define your own output images here as needed




    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn  = new float[(size_t)w*h*nc];
    size_t imageBytes = (size_t)w*h*nc*sizeof(float);

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];




    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);

    dim3 block(32, 16);
    dim3 grid = make_grid(dim3(w, h, 1), block);

    Timer timer; timer.start();
    float *d_U, *d_F, *d_G;
    hipMalloc(&d_U, imageBytes);
    hipMalloc(&d_F, imageBytes);
    hipMalloc(&d_G, (size_t)w*h*sizeof(float));
    hipMemcpy(d_U, imgIn, imageBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_F, imgIn, imageBytes, hipMemcpyHostToDevice);

    for (int n = 0; n < N; n++) {
        diffusivity<<< grid, block >>>(d_U, d_G, w, h, nc, epsilon);
        sor_update<<< grid, block >>>(d_U, d_F, d_G, w, h, nc, lambda, 0);
        sor_update<<< grid, block >>>(d_U, d_F, d_G, w, h, nc, lambda, 1);
    }

    hipMemcpy(imgOut, d_U, imageBytes, hipMemcpyDeviceToHost);
    hipFree(d_U);
    hipFree(d_F);
    hipFree(d_G);
    timer.end();  float t = timer.get();  // elapsed time in seconds
    cout << "time: " << t*1000 << " ms" << endl;






    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, imgOut);
    showImage("Output", mOut, 100+w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif




    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



