// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Winter Semester 2013/2014, March 3 - April 4
// ###
// ###
// ### Evgeny Strekalovskiy, Maria Klodt, Jan Stuehmer, Mohamed Souiai
// ###
// ###
// ###


#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;



// cuda error checking
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        exit(1);
    }
}


__device__ float square(float x)
{
    return x * x;
}

__global__ void square_array(float *arr, int n)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n)
        arr[idx] = square(arr[idx]);
}

int main(int argc,char **argv)
{
    // alloc and init input arrays on host (CPU)
    int n = 10;
    float *a = new float[n];
    for(int i=0; i<n; i++) a[i] = i;

    // CPU computation
    for(int i=0; i<n; i++)
    {
        float val = a[i];
        val = val*val;
        a[i] = val;
    }

    // print result
    cout << "CPU:"<<endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << endl;
    cout << endl;
    


    // GPU computation
    // reinit data
    for(int i=0; i<n; i++) a[i] = i;

    float *d_a;
    hipMalloc(&d_a, n*sizeof(float));
    hipMemcpy(d_a, a, n*sizeof(float), hipMemcpyHostToDevice);

    dim3 block(32);
    dim3 grid((n + block.x - 1) / block.x);
    square_array<<<grid, block>>>(d_a, n);

    hipMemcpy(a, d_a, n*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_a);

    // print result
    cout << "GPU:" << endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << endl;
    cout << endl;

    // free CPU arrays
    delete[] a;
}



